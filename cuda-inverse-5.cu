

#include <hip/hip_runtime.h>
#include <stdio.h> //file i/o commands
#include <stdlib.h>
#include <math.h>
#include <sys/stat.h> //checking file types of file descriptors
#include <sys/types.h> //fstat, other file i/o, wait
#include <sys/resource.h> //for wait
#include <sys/wait.h> //for eait
#include <getopt.h> //for getopt
#include <unistd.h> //pipe command, parsing command line stuff, pause, forking
#include <stdlib.h> //malloc and stuff
#include <errno.h> //for the errno checking
#include <fcntl.h> //for O_CONSTANTS
#include <string.h> //fprintf, etc.

#include <sys/time.h>
struct timeval start_time, stop_time;
int    elapsed;


/* --------------------------------------------------------------
   Help function for your CUDA kernel to print the
   input matrices.


   Make sure that ONLY 1 thread calls this printMatrixPair( )
   function ! (or else, you will see many many outputs !!!)

   In other words, use:

     if ( myUniqueID == 0 )
        printMatrixPair(A,C,n);

   *** If you DID use this help function to debug, you must REMOVE 
   *** the added code when you turn in the program 
   *** (I will test with large matrices and it will cause a lot 
   *** of printing !!!
   ------------------------------------------------------------- */
__device__
void printMatrixPair( float *A, float *C, int n )
{
   for (int i = 0; i < n; i++ )
   {
      for (int j = 0; j < n; j++ )
      {
         printf("%6.2f ", A[i*n+j]);
      }
      printf("  |  ");
      for (int j = 0; j < n; j++ )
      {
         printf("%6.2f ", C[i*n+j]);
      }

      printf("\n");
   }
}


/* =======================================================================
   TODO: write a CUDA kernel to compute the inverse of matrix A

   inverse(A, C, N):

      input:  A = an NxN matrix that you need to find the inverse
                  When the inverse( ) function completes, A will
                  contains the identity matrix
              C = initially contains the identity matrix
                  When the inverse( ) function completes, C will
                  contains the inverse of A
              N = #row (and # columns) in A and C
   ======================================================================== */
__global__ void inverse( float *A, float *C, int N)
{
        /* ==========================================================
           R = a variable that goes through each row of the matrix
           ========================================================== */
        printf("block id (%d, %d), thread id (%d, %d)\n",blockIdx.x, blockIdx.y, threadIdx.x,threadIdx.y);
        for (int R = 0; R < N; R++)
        {
        float factor = A[R*N + R];	// Divide each element by A(R,R) i.e. the diagonal 
    
        int col = threadIdx.x;
        
        A[R*N+col] = A[R*N+col]/factor; //divide A(:,col) by its diagonal
        C[R*N+col] = C[R*N+col]/factor; 
        
        __syncthreads( );
        /* =========================================================
            Make a column of 0 values in column R using the row "R"
            ========================================================= */
        int T = threadIdx.x;
         if (T == R)
         {
               // Do nothing to row "R"
         }
         else
         {
               // Multiply factor: A[T][R] == A(T,R)
               float f = A[T*N+R];		
               /* -------------------------------------
               Add  -f*row(R) to row(i) 
               ------------------------------------- */
               for(int j=0; j < N; j++){
                  A[T*N+j] -= f*A[R*N+j]; //add -A[T,R]*A[R,col] to A[T,col]
                  C[T*N+j] -= f*C[R*N+j];
               }
         }
         __syncthreads( );
    }
      
}

 




   /* =============================================================
      Hint:
         0. Main() has spawn N thread

      inverse( ) must:

         1. find the thread ID of the CUDA thread that runs this kernel
            Assume this is thread T

         2. Thread T must process every row in the matrices A and C as
            follows:

              (Processing of row R):
              Step 1: Normalize row R

              The work in step 1 is divided as follows:

                  Thread T computes: A[R][T] = A[R][T]/A[R][R]
                                     C[R][T] = C[R][T]/A[R][R]

              Step 2. Use row R to create a column of 0 values
                      in the column R

              The work in step 2 is divided as follows:

                  Thread T adds  -A[T][R]*row(R) to row(T)
      ============================================================= */





void printMatrix( float *A, int n )
{
   for (int i = 0; i < n; i++ )
   {
      for (int j = 0; j < n; j++ )
      {
         printf("%6.2f ", A[i*n+j]);
      }
      printf("\n");
   }
}



int main(int argc, char *argv[])
{
  if ( argc <= 1 )
  {
     printf("Usage: %s N (NxN matrix inversion)\n\n", argv[0]);
     exit(1);
  }

  int N = atoi( argv[1] );

  float *A, *C, *A_org;

  /* ====================================
     Allocate arrays
     ==================================== */
  hipMallocManaged(&A, N*N*sizeof(float));
  A_org = (float*) calloc(N*N, sizeof(float));
  hipMallocManaged(&C, N*N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
     for (int j = 0; j < N; j++)
     {
        A[i*N+j] = rand()/1000000000.0;
        A_org[i*N+j] = A[i*N+j];

        if ( i == j )
           C[i*N+j] = 1.0;
        else
           C[i*N+j] = 0.0;
     }

   printf("Input matrix: printing to input.txt\n");
   fflush(stdout);
   remove("input.txt");
   int fp = open("input.txt",O_WRONLY|O_CREAT, S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
   dup2(1,5); //copy stdout to a new fp
   dup2(fp,1); //copy fp into stdout
   printMatrix( A, N );
   dup2(5,1); //get rid of stdout
   close(fp); //close the file pointer

  gettimeofday(&start_time, NULL);   // Record current sys time as start_time
  /* ========================================================
     CPU algorithm for matrix inversion using Kramer's rule
     ======================================================== */
  // ==================================================================
  // Run kernel on the GPU using 1 block, N thread/per block
  inverse<<<1, N>>>( A, C, N);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  // ==================================================================


   gettimeofday(&stop_time, NULL);   // Record current sys time as stop_time

   elapsed = (stop_time.tv_sec*1000000 + stop_time.tv_usec) -
                (start_time.tv_sec*1000000 + start_time.tv_usec);
   printf("Elasped time = %d micro secs\n", elapsed);

   printf("Matrix A: printing to A_after.txt\n");
   fflush(stdout);
   remove("A_after.txt");
   fp = open("A_after.txt",O_WRONLY|O_CREAT, S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
   dup2(1,5); //copy stdout to a new fp
   dup2(fp,1); //copy fp into stdout
   printMatrix( A_org, N );
   dup2(5,1); //get rid of stdout
   close(fp); //close the file pointer

   printf("Inverse matrix: printing to inverse_mat.txt\n");
   fflush(stdout);
   remove("inverse_mat.txt");
   fp = open("inverse_mat.txt",O_WRONLY|O_CREAT, S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
   dup2(1,5); //copy stdout to a new fp
   dup2(fp,1); //copy fp into stdout
   printMatrix( C, N );
   dup2(5,1); //get rid of stdout
   close(fp); //close the file pointer

   /* ====================================================
      Check if inverse is correct
      ==================================================== */
  int numErr = 0;

  for (int i = 0; i < N; i++)
     for (int j = 0; j < N; j++)
     {
        // Compute i,j-th element of A * Ainv

        float sum = 0;

        for ( int k = 0; k < N; k++ )
           sum += A_org[i*N+k] * C[k*N+j];

//      printf("Mult[%d][%d] = %f\n", i, j, sum);

        if ( i == j )
        {
           if ( fabs(sum - 1.0) > 0.15 )  numErr++;
        }
        else
        {
           if ( fabs(sum - 0.0) > 0.15 )  numErr++;
        }
     }

   if ( numErr > 0 )
      printf("**** Inverse is NOT correct !\n");
   else
      printf("Inverse is correct !\n");

   // Free memory
   hipFree(A);
   hipFree(C);
   free(A_org);

  return 0;
}

